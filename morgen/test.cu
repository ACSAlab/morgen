#include "hip/hip_runtime.h"
/*
 *   For testing
 *
 *   Copyright (C) 2013-2014 by
 *   Yichao Cheng        onesuperclark@gmail.com
 *
 *   This program is free software; you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation; either version 2 of the License, or
 *   (at your option) any later version.
 *
 *   This program is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 */



#include <morgen/graph/csr_graph.cuh>
#include <morgen/graph/gen/mine.cuh>
#include <morgen/graph/gen/dimacs.cuh>
#include <morgen/graph/gen/coo.cuh>
#include <morgen/bfs/bitmask.cu>
#include <morgen/bfs/queue.cu>
#include <morgen/bfs/hash.cu>
#include <morgen/bfs/serial.cu>
#include <morgen/utils/command_line.cuh>
#include <morgen/utils/random_node.cuh>
#include <morgen/utils/utilizing_efficiency.cuh>


using namespace morgen;


void usage() {
    printf("\ntest <graph> <bfs type> [--device=<device index>] "
            "[--slots=<number of slots>] [--outdegree] [--distribution] [--workset]"
            "[--src=<source idx>] [--instrument] [--random_source] "
            "[--group_size=<group size>]\n"
            "\n"
            "<graph>\n"
            "  tiny: tiny graph for debugging\n"
            "  fla: Florida Road map\n"
            "  mesh: 6-point 2D mesh\n"
            "  kkt: Optimal power flow, nonlinear optimization (KKT)\n"
            "  copaper: CopaperCiteSeer\n"
            "  audi: symmetric rb matrix\n"
            "  rmat1: random small world graph (n=5M  m=60M)\n"
            "  rmat2: random small world graph (n=2M  n=100M)\n"
            "  amazon: Amazon product co-buying\n"
            "  random1: Erdos-Renyi or uniformly random graph (n=5M n=60M)\n"
            "  random2: Erdos-Renyi or uniformly random graph (n=2M n=100M)\n"
            "  eco: circuit theory applied to animal/gene flow\n"
            "  thermal: FEM 3D nonlinear thermal problem, 8-node bricks as volume elements\n"
            "  livejournal: LiveJournal's social network\n"
            "\n");
}


void check_open(FILE *fp, char *filename) {
    if (!fp) {
        fprintf(stderr, "cannot open file: %s\n", filename);
        exit(1);
    }
}


int main(int argc, char **argv) {
    

    typedef int VertexId;
    typedef int SizeT;
    typedef int Value;



    /*********************************************************************
     * Commandline parsing
     *********************************************************************/
    util::CommandLineArgs args(argc, argv);
    // 0: prog   1: graph    2: bfs_type
    if ((argc < 3) || args.CheckCmdLineFlag("help")) {
        usage();
        return 1;
    }

    std::string graph = argv[1];
    std::string bfs_type = argv[2];


    printf("================================================================\n");
    printf("[opt] Graph:\t\t%s\n", graph.c_str());


    /*********************************************************************
     * Parse arguments and display them on the screen
     *********************************************************************/

    // --outdegree=<log>|<uniform> : print out degrees of the graph?
    bool display_outdegree_uniform = false;
    bool display_outdegree_log = false;

    std::string outdegree_str;
    args.GetCmdLineArgument("outdegree", outdegree_str);

    if (outdegree_str.compare("log") == 0) {
        display_outdegree_log = true;
    } else if (outdegree_str.compare("uniform") == 0) {
        display_outdegree_uniform = true;
    }
    
    if (display_outdegree_uniform) {
        printf("Display outdegree: \tuniform\n");
	} else if (display_outdegree_log){
        printf("Display outdegree: \tuniform\n");
    } else {
        printf("Display outdegree: \t\tNo\n");
    }

    // --distribution : print the edge distribution each level?
    bool display_distribution = args.CheckCmdLineFlag("distribution");
    printf("Display distribution?\t\t%s\n", (display_distribution ? "Yes" : "No"));

    // --workset :
    bool display_workset = args.CheckCmdLineFlag("workset");
    printf("Display workset?\t\t%s\n", (display_workset ? "Yes" : "No"));

    // --metrics :
    bool display_metrics = args.CheckCmdLineFlag("metrics");
    printf("Display metrics?\t\t%s\n", (display_metrics ? "Yes" : "No"));

    // --warp_map :
    bool warp_mapped = args.CheckCmdLineFlag("warp_map");
    printf("Warp mapping?\t\t%s\n", (warp_mapped ? "Yes" : "No"));

    // --instrument : whether instrument each frontier
    bool instrument = args.CheckCmdLineFlag("instrument");
    printf("Instrument?\t\t%s\n", (instrument ? "Yes" : "No"));

    // --source=<source node ID> | <random>
    VertexId source = 0;
    std::string src_str;
    bool randomized_source = false;
    args.GetCmdLineArgument("source", src_str);
    if (src_str.compare("random") == 0) {
        randomized_source = true;
        printf("Source node:\trandomized\n");
    } else {
        args.GetCmdLineArgument("source", source);
        printf("Source node:\t%d\n", source);
    }

    // --slots=<number of slots>
    int slots = 0;
    args.GetCmdLineArgument("slots", slots);
    printf("Slot number:\t\t%d\n", slots);

    // --block_size=<block size>
    int block_size = 256;
    args.GetCmdLineArgument("block_size", block_size);
    printf("BLock size(threads):\t%d\n", block_size);

    // --group_size=<group size>
    int group_size = 32;
    args.GetCmdLineArgument("group_size", group_size);
    printf("Group size(threads):\t%d\n", group_size);


    graph::CsrGraph<VertexId, SizeT, Value> ga;


    /*********************************************************************
     * Build the graph from a file
     *********************************************************************/
    FILE *fp;

    if (graph == "tiny") {
        fp = fopen(getenv("TINY_GRAPH"), "r");
        check_open(fp, "tiny");
        if (graph::gen::myGraphGen<VertexId, SizeT, Value>(fp, ga) != 0) return 1;

    } else if (graph == "fla") {
        fp = fopen(getenv("FLA_GRAPH"), "r");
        check_open(fp, "fla");
        if (graph::gen::myGraphGen<VertexId, SizeT, Value>(fp, ga) != 0) return 1;

    } else if (graph == "mesh") {
    
        fp = fopen(getenv("MESH_GRAPH"), "r");
        check_open(fp, "mesh");
        if (graph::gen::myGraphGen<VertexId, SizeT, Value>(fp, ga) != 0) return 1;

    } else if (graph == "rmat1") {
    
        fp = fopen(getenv("RMAT1_GRAPH"), "r");
        check_open(fp, "rmat1");
        if (graph::gen::myGraphGen<VertexId, SizeT, Value>(fp, ga) != 0) return 1;

    } else if (graph == "rmat2") {
    
        fp = fopen(getenv("RMAT2_GRAPH"), "r");
        check_open(fp, "rmat2");
        if (graph::gen::cooGraphGen<VertexId, SizeT, Value>(fp, ga) != 0) return 1;

    } else if (graph == "random1") {
    
        fp = fopen(getenv("RANDOM1_GRAPH"), "r");
        check_open(fp, "random1");
        if (graph::gen::cooGraphGen<VertexId, SizeT, Value>(fp, ga) != 0) return 1;

    } else if (graph == "random2") {
    
        fp = fopen(getenv("RANDOM2_GRAPH"), "r");
        check_open(fp, "random2");
        if (graph::gen::cooGraphGen<VertexId, SizeT, Value>(fp, ga) != 0) return 1;

    } else if (graph == "kkt") {

        fp = fopen(getenv("KKT_GRAPH"), "r");
        check_open(fp, "kkt");
        if (graph::gen::dimacsGraphGen<VertexId, SizeT, Value>(fp, ga) != 0) return 1;

    } else if (graph == "copaper") {

        fp = fopen(getenv("COPAPER_GRAPH"), "r");
        check_open(fp, "copaper");
        if (graph::gen::dimacsGraphGen<VertexId, SizeT, Value>(fp, ga) != 0) return 1;

    } else if (graph == "audi") {

        fp = fopen(getenv("AUDI_GRAPH"), "r");
        check_open(fp, "audi");
        if (graph::gen::dimacsGraphGen<VertexId, SizeT, Value>(fp, ga) !=0 ) return 1;

    } else if (graph == "amazon") {

        fp = fopen(getenv("AMAZON_GRAPH"), "r");
        check_open(fp, "amazon");
        if (graph::gen::cooGraphGen<VertexId, SizeT, Value>(fp, ga) != 0) return 1;
    
    } else if (graph == "thermal") {

        fp = fopen(getenv("THERMAL_GRAPH"), "r");
        check_open(fp, "thermal");
        if (graph::gen::dimacsGraphGen<VertexId, SizeT, Value>(fp, ga) != 0) return 1;

    } else if (graph == "eco") {

        fp = fopen(getenv("ECO_GRAPH"), "r");
        check_open(fp, "eco");
        if (graph::gen::dimacsGraphGen<VertexId, SizeT, Value>(fp, ga) != 0) return 1;

    } else if (graph == "livejournal") {

        fp = fopen(getenv("LIVE_GRAPH"), "r");
        check_open(fp, "livejournal");
        if (graph::gen::cooGraphGen<VertexId, SizeT, Value>(fp, ga) != 0) return 1;

    } else {
        fprintf(stderr, "no graph is specified\n");
        return 1;
    }

    /*********************************************************************
     * Display
     *********************************************************************/

    // Graph Information display(not verbose)
    ga.printInfo(false); 

    if (display_outdegree_log) 
        ga.printOutDegreesLog();

	if (display_outdegree_uniform) 
        ga.printOutDegreesUniform();
		
    if (display_distribution || display_workset) 
        bfs::BFSGraph_serial<VertexId, SizeT, Value>(
            ga,
            (VertexId) 0, 
            instrument, 
            display_distribution,
            display_workset);

    if (display_metrics)
        util::displayUtilizingEfficiency(ga);


    /*********************************************************************
     * Decide which node to start from
     *********************************************************************/

    if (randomized_source)
        source = util::randomNode(ga.n);
    else
        source = source % ga.n;

    printf("Traversing from %d\n", source);    


    /*********************************************************************
     * Traversing
     *********************************************************************/
    if (bfs_type == "serial") {

        bfs::BFSGraph_serial<VertexId, SizeT, Value>(
            ga, 
            source,
            instrument,
            display_distribution,
            display_workset);

    } else if (bfs_type == "bitmask") {

        bfs::BFSGraph_gpu_bitmask<VertexId, SizeT, Value>(
            ga,
            source,
            instrument);

    } else if (bfs_type == "queue") {

        bfs::BFSGraph_gpu_queue<VertexId, SizeT, Value>(
            ga,                                            
            source,
            instrument,
            block_size,
            warp_mapped,
            group_size);

    } else if (bfs_type == "hash") {

        bfs::BFSGraph_gpu_hash<VertexId, SizeT, Value>(
            ga,
            source, 
            slots, 
            instrument);

    } else {
        fprintf(stderr, "no traverse type is specified. exit quietly\n");
    }


    fclose(fp);
    ga.del();
    return 0;
}