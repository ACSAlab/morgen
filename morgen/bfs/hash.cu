#include "hip/hip_runtime.h"

/*
 *   The breadth-first search algorithm
 *
 *   Copyright (C) 2013-2014 by
 *   Cheng Yichao        onesuperclark@gmail.com
 *
 *   This program is free software; you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation; either version 2 of the License, or
 *   (at your option) any later version.
 *
 *   This program is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 */


#pragma once

#include <morgen/utils/macros.cuh>
#include <morgen/utils/timing.cuh>
#include <morgen/utils/list.cuh>
#include <morgen/workset/hash.cuh>

#include <hip/hip_runtime_api.h>


namespace morgen {

namespace bfs {


/**
 * This is a fixed thread-mapping kernel for hashe-based workset
 * The workset of current level is processed in one kernal launch
 */
template<typename VertexId, typename SizeT, typename Value>
__global__ void
BFSKernel(SizeT     *row_offsets,
          VertexId  *column_indices,
          VertexId  *workset_from,
          SizeT     slot_num_from,
          SizeT     *slot_offsets_from,
          SizeT     *slot_sizes_from,
          SizeT     *workset_to,
          SizeT     slot_num_to,
          SizeT     *slot_offsets_to,
          VertexId  *slot_sizes_to,
          Value     *levels,
          Value     curLevel,
          int       *visited)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Initially, clean the workset to empty(each slot counter = 0)
    if (tid < slot_num_to) {
        slot_sizes_to[tid] = 0;
    }

    __syncthreads();


    // tid  0 1 2 3 4 5  <- accord threads num with the logical size   
    //  0   a b c d e f
    //  1   g h i j
    //  2   k l m n o
    //  3   p q
    for (int i = 0; i < slot_num_from; i++) {

        if (tid < slot_sizes_from[i]) {

            VertexId outNode = workset_from[slot_offsets_from[i] + tid];
            SizeT outEdgeFirst = row_offsets[outNode];
            SizeT outEdgeLast = row_offsets[outNode+1];

            // serial expansion
            for (SizeT edge = outEdgeFirst; edge < outEdgeLast; edge++) {

                VertexId inNode = column_indices[edge];

                // resolve concurrent discovery through atomical operations
                int old = atomicExch( (int*)&visited[inNode], 1 );

                if (old == 0) { 

                    levels[inNode] = curLevel + 1;

                    // hash the pos by inNode id
                    int hash = inNode % slot_num_to;

                    // exclusively get the writing position within the slot
                    SizeT pos= atomicAdd( (SizeT*) &(slot_sizes_to[hash]), 1 );
                    workset_to[slot_offsets_to[hash] + pos] = inNode;
                }
            }   
        }
    }   
}


/**
 * Each vertex(u) in worksetFrom is assigned with a group of threads.
 * Then each thead within a group processes one of u's neigbors
 * at a time. All threads process vertices in SIMD manner.
 *
 * Assume GROUP_S = 32
 * If u has a neigbor number more than 32, each thead within a group will 
 * iterate over them stridedly. e.g. thread 1 will process 1st, 33th, 65th... 
 * vertex in the neighbor list, thread 2 will process 2nd, 34th, 66th...
 */
template<typename VertexId,
         typename SizeT,
         typename Value>
__global__ void
BFSKernel_warp_mapped(SizeT     *row_offsets,
                      VertexId  *column_indices,
                      VertexId  *worksetFrom,
                      SizeT     *sizeFrom,
                      VertexId  *worksetTo,
                      SizeT     *sizeTo,
                      Value     *levels,
                      Value     curLevel,
                      int       *visited,
                      int       group_size,
                      int       group_per_block)
{



    int tid = blockIdx.x * blockDim.x + threadIdx.x;


    int group_offset = tid % group_size;
    int group_id     = tid / group_size;

    // writing to an empty buffer
    if (tid == 0) *sizeTo = 0;

    __syncthreads();


    // Each group has use a variable to record the total work
    // amount(neigbors) that belongs to that group
    // groups/block = thread per block / group size
    // The size is allocated dynamically
    volatile __shared__ SizeT edge_first[256];
    volatile __shared__ SizeT edge_last[256];


    // Since the workset can easily exceed 65536, we just let grouped-threads
    // iterate over a large workset
    for (int g = group_id; g < *sizeFrom; g += group_per_block * gridDim.x) {

        //if (g % 1024 == 0 && group_offset == 0)
        //    printf("I am group %d, size: %d, my next: %d\n", g, *sizeFrom, g+group_per_block * gridDim.x);


        // First thread in the group do this job read out info 
        // from global mem to local mem
        if (group_offset == 0) {

            VertexId outNode = worksetFrom[g];
            levels[outNode] = curLevel;
            edge_first[group_id % group_per_block] = row_offsets[outNode];
            edge_last[group_id % group_per_block] = row_offsets[outNode+1];
        }

        __syncthreads();
    
        // in case the neighbor number > warp size
        for (SizeT edge = edge_first[group_id % group_per_block] + group_offset;
             edge < edge_last[group_id % group_per_block];
             edge += group_size)
        {
            
            VertexId inNode = column_indices[edge];

            int old = atomicExch( (int*)&visited[inNode], 1 );

            if (old == 0) { 
                // fine-grained allocation
                SizeT pos= atomicAdd( (SizeT*) &(*sizeTo), 1 );
                worksetTo[pos] = inNode;
            }
        }
    }
}


template<typename VertexId, typename SizeT, typename Value>
void BFSGraph_gpu_hash(
    const graph::CsrGraph<VertexId, SizeT, Value> &g, 
    VertexId source, 
    int slots,
    bool instrument = false)
{

    if (slots > 0) {
        printf("Slots = %d\n", slots);
    }
    else {
        printf("Slots should be a positive number\n");
        return;
    }

    // To make better use of the workset, we create two.
    // Instead of creating a new one everytime in each BFS level,
    // we just expand vertices from one to another
    workset::NaiveHash<VertexId, SizeT> workset1(g.n, slots);
    workset::NaiveHash<VertexId, SizeT> workset2(g.n, slots);


    // Initalize auxiliary list
    util::List<Value, SizeT> levels(g.n);
    levels.all_to((Value) MORGEN_INF);


    // visitation list: 0 for unvisited
    util::List<int, SizeT> visited(g.n);
    visited.all_to(0);


    // traverse from source node
    workset1.insert(source);   
    levels.set(source, 0);
    visited.set(source, 1);

    SizeT worksetSize = 1;
    SizeT lastWorksetSize = 0;
    SizeT actualWorksetSize = 1; 
    SizeT lastActualWorksetSize = 0;

    Value curLevel = 0;

    // kernel configuration
    int blockNum = 16;
    int blockSize = 256;


    printf("GPU hashed bfs starts... \n");  
    

    if (instrument)
        printf("level\tslot_size\tfrontier_size\tratio\ttime\n");

    float total_millis = 0.0;

    while (worksetSize > 0) {

        lastWorksetSize = worksetSize;
        lastActualWorksetSize = actualWorksetSize;

        // In hashed version,  the worksetSize is the logical size
        // of the hash table(smallest among the slot sizes)
        blockNum = (worksetSize % blockSize == 0 ? 
            worksetSize / blockSize :
            worksetSize / blockSize + 1);


        // kick off timer first
        util::GpuTimer gpu_timer;
        gpu_timer.start();

        if (curLevel % 2 == 0) 
        {

            // call kernel with device pointers
            BFSKernel<<<blockNum, blockSize>>>(g.d_row_offsets,
                                               g.d_column_indices,
                                               workset1.d_elems,
                                               workset1.slot_num,
                                               workset1.d_slot_offsets,
                                               workset1.d_slot_sizes,                                    
                                               workset2.d_elems,
                                               workset2.slot_num,
                                               workset2.d_slot_offsets,
                                               workset2.d_slot_sizes,
                                               levels.d_elems,
                                               curLevel,     
                                               visited.d_elems);

            if (util::handleError(hipDeviceSynchronize(), "BFSKernel failed ", __FILE__, __LINE__)) break;


            worksetSize = workset2.max_slot_size();
            actualWorksetSize = workset2.sum_slot_size();
         } else {

            BFSKernel<<<blockNum, blockSize>>>(g.d_row_offsets,
                                               g.d_column_indices,
                                               workset2.d_elems,
                                               workset2.slot_num,
                                               workset2.d_slot_offsets,
                                               workset2.d_slot_sizes,                                    
                                               workset1.d_elems,
                                               workset1.slot_num,
                                               workset1.d_slot_offsets,
                                               workset1.d_slot_sizes,
                                               levels.d_elems,
                                               curLevel,     
                                               visited.d_elems);

            if (util::handleError(hipDeviceSynchronize(), "BFSKernel failed ", __FILE__, __LINE__)) break;

            
            worksetSize = workset1.max_slot_size();
            actualWorksetSize = workset1.sum_slot_size();

         }

         // timer end
         gpu_timer.stop();
         float mapping_efficiency = (float) lastActualWorksetSize / (lastWorksetSize * slots);
         total_millis += gpu_timer.elapsedMillis();
         if (instrument) printf("%d\t%d\t%d\t%.3f\t%f\n", curLevel, lastWorksetSize, lastActualWorksetSize, mapping_efficiency, gpu_timer.elapsedMillis());

         curLevel += 1;

    }
    
    printf("GPU hashed bfs terminates\n");
    float billion_edges_per_second = (float)g.m / total_millis / 1000000.0;
    printf("Time(s):\t%f\nSpeed(BE/s):\t%f\n", total_millis / 1000.0, billion_edges_per_second);

    levels.print_log();

    levels.del();
    visited.del();
    workset1.del();
    workset2.del();
    
}


} // BFS
} // Morgen