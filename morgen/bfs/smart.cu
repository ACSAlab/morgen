#include "hip/hip_runtime.h"

/*
 *   The breadth-first search algorithm
 *
 *   Copyright (C) 2013-2014 by
 *   Cheng Yichao        onesuperclark@gmail.com
 *
 *   This program is free software; you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation; either version 2 of the License, or
 *   (at your option) any later version.
 *
 *   This program is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 */


#pragma once

#include <morgen/utils/macros.cuh>
#include <morgen/utils/timing.cuh>
#include <morgen/utils/list.cuh>
#include <morgen/utils/log.cuh>
#include <morgen/workset/hash.cuh>

#include <hip/hip_runtime_api.h>


namespace morgen {

namespace bfs {


/**
 * This is a fixed thread-mapping kernel for hashe-based workset
 * The workset of current level is processed in one kernal launch
 */
template<typename VertexId, 
         typename SizeT,
         typename Value,
         bool ORDERED>
__global__ void
BFSKernel_smart_thread_map(
  SizeT     *row_offsets,
  VertexId  *column_indices,
  VertexId  *workset_from,
  SizeT     *slot_offsets_from,
  SizeT     *slot_sizes_from,
  int       slot_id_from,
  Value     *levels,
  Value     curLevel,
  int       *visited,
  int       *update)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;


    if (tid < slot_sizes_from[slot_id_from]) {

        VertexId outNode = workset_from[slot_offsets_from[slot_id_from] + tid];
        SizeT outEdgeFirst = row_offsets[outNode];
        SizeT outEdgeLast = row_offsets[outNode+1];

        for (SizeT e = outEdgeFirst; e < outEdgeLast; e++) {
            VertexId inNode = column_indices[e];
            Value level = curLevel + 1;

            if (ORDERED) {
                if (visited[inNode] == 0) {
                    levels[inNode] = level;
                    update[inNode] = 1;
                }
            } else {
                if (levels[inNode] > level) {
                    levels[inNode] = level;
                    update[inNode] = 1;
                }
            }

       }
    }   
    
    
}



template<typename VertexId, 
         typename SizeT, 
         typename Value,
         bool ORDERED>
__global__ void
BFSKernel_smart_group_map(
  SizeT     *row_offsets,
  VertexId  *column_indices,
  VertexId  *workset_from,
  SizeT     *slot_offsets_from,
  SizeT     *slot_sizes_from,
  int       slot_id_from,
  Value     *levels,
  Value     curLevel,
  int       *visited,
  int       group_size,
  float     group_per_block,
  int       *update)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int group_offset = tid % group_size;
    int group_id     = tid / group_size;


    // group_per_block * gridDim.x = how many groups of threads are spawned 
    for (int g = group_id; g < slot_sizes_from[slot_id_from]; g += group_per_block * gridDim.x) {

        VertexId outNode = workset_from[slot_offsets_from[slot_id_from] + g];
        SizeT edgeFirst = row_offsets[outNode];
        SizeT edgeLast = row_offsets[outNode+1];

        // serial expansion
        for (SizeT edge = edgeFirst + group_offset; edge < edgeLast; edge += group_size) 
        {

            VertexId inNode = column_indices[edge];
            Value level = curLevel + 1;

            if (ORDERED) {
                if (visited[inNode] == 0) {
                    levels[inNode] = level;
                    update[inNode] = 1;
                }
            } else {
                if (levels[inNode] > level) {
                    levels[inNode] = level;
                    update[inNode] = 1;
                }
            }
        } // edge loop
    }
}


/**
 * use update[] to mask activated[]
 */
template<typename VertexId, typename SizeT>
__global__ void
BFSKernel_smart_gen_workset(
    SizeT     max_size,
    SizeT     *row_offsets,
    VertexId  *column_indices,
    int       *visited,
    int       *update,
    int       *outdegrees,
    VertexId  *workset_to,
    SizeT     *slot_offsets_to,
    VertexId  *slot_sizes_to)
{
    int tid =  blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < max_size) {

        if (update[tid] == 1) {

            update[tid] = 0;     // clear after activating
            visited[tid] = 1;

            int hash = outdegrees[tid];
            if (hash >= 0) {
                SizeT pos = atomicAdd( (SizeT*) &(slot_sizes_to[hash]), 1 );
                workset_to[slot_offsets_to[hash] + pos] = tid;
            }
        }
    }
}



template<typename VertexId, typename SizeT, typename Value>
void BFSGraph_gpu_smart(
    const graph::CsrGraph<VertexId, SizeT, Value> &g, 
    VertexId source, 
    const util::Stats<VertexId, SizeT, Value> &stats,
    bool instrument,
    int block_size,
    bool unordered)
{


    // To make better use of the workset, we create two.
    // Instead of creating a new one everytime in each BFS level,
    // we just expand vertices from one to another
    workset::Hash<VertexId, SizeT, Value>  workset[] = {
        workset::Hash<VertexId, SizeT, Value>(stats),
        workset::Hash<VertexId, SizeT, Value>(stats),
    };





    // create a outdegree table first
    // outdegree:     0  (0,1]  (1, 2]  (2, 4]   (4, 8]   (8, 16]
    // altered       -1   0      1       2       3        4       
    util::List<Value, SizeT> outdegreesLog(g.n);
    for (SizeT i = 0; i < g.n; i++) {
        SizeT outDegree = g.row_offsets[i+1] - g.row_offsets[i];
        if (outDegree == 0) 
            outdegreesLog.elems[i] = -1;
        else if (outDegree > 0 && outDegree <= 1)
            outdegreesLog.elems[i] = 0;
        else if (outDegree > 1 && outDegree <= 2)
            outdegreesLog.elems[i] = 1;
        else if (outDegree > 2 && outDegree <= 4)
            outdegreesLog.elems[i] = 2;
        else if (outDegree > 4 && outDegree <= 8)
            outdegreesLog.elems[i] = 3;
        else if (outDegree > 8 && outDegree <= 16)
            outdegreesLog.elems[i] = 4;
        else 
            outdegreesLog.elems[i] = 5;
    }
    outdegreesLog.transfer();

    // use to select between two worksets
    // src:  workset[selector]
    // dest: workset[selector ^ 1]
    int selector = 0;

    // Initalize auxiliary list
    util::List<Value, SizeT> levels(g.n);
    levels.all_to((Value) MORGEN_INF);

    // visitation list: 0 for unvisited
    util::List<int, SizeT> visited(g.n);
    visited.all_to(0);

    util::List<int, SizeT> update(g.n);
    update.all_to(0);


    // traverse from source node
    workset[0].insert(outdegreesLog.elems[source], source);   
    levels.set(source, 0);
    visited.set(source, 1);
    SizeT worksetSize = 1;
    SizeT lastWorksetSize = 0;
    Value curLevel = 0;
    //int accumulatedBlocks = 0;


    printf("GPU topology-aware bfs starts... \n");  

    if (instrument) printf("level\tslot_size\tfrontier_size\tratio\ttime\n");
    float total_millis = 0.0;


    while (worksetSize > 0) {

        lastWorksetSize = worksetSize;

        // kick off timer first
        util::GpuTimer gpu_timer;
        gpu_timer.start();

        workset[selector ^ 1].clear_slot_sizes();

        // expand edges slot by slot
        // i:           0  1  2  3  4   5   6...
        // group_size:  1  2  4  8  16  32  32
        for (int i = 0; i < workset[selector].slot_num; i++) {

 

            int partialWorksetSize = workset[selector].slot_sizes[i];

            // skip the empty slot
            if (partialWorksetSize== 0) continue;

            // decide which mapping strategy to be used according to i
            int group_size = 0;
            switch (i) {
                case 0: group_size = 1; break;
                case 1: group_size = 2; break;
                case 2: group_size = 4; break;
                case 3: group_size = 8; break;
                case 4: group_size = 16; break;
                default: group_size = 32; 
                /*
                case 5: group_size = 32; break;
                case 6: group_size = 64; break;
                case 7: group_size = 128; break;
                case 8: group_size = 256;break;
                case 9: group_size = 512; break;
                case 10: group_size = 1024;break;
                case 11: group_size = 2048; break;
                case 12: group_size = 4096; break;
                case 13: group_size = 8192; break;
                case 14: group_size = 16384; break;
                case 15: group_size = 32768; break;
                default: fprintf(stderr, "out of control!!\n"); return;*/
            }


            // will be used in the kernel
            float group_per_block = (float)block_size / group_size;

            // In hashed version,  the worksetSize is the logical size
            // of the hash table(smallest among the slot sizes)
            int blockNum = ((partialWorksetSize * group_size) % block_size == 0 ? 
                partialWorksetSize * group_size / block_size :
                partialWorksetSize * group_size / block_size + 1);

            // safe belt: grid width has a limit of 65535
            if (blockNum > 65535) blockNum = 65535;


            if (group_size == 1) {

                BFSKernel_smart_thread_map<VertexId, SizeT, Value, true><<<blockNum, block_size>>>(
                    g.d_row_offsets,
                    g.d_column_indices,
                    workset[selector].d_elems,
                    workset[selector].d_slot_offsets,
                    workset[selector].d_slot_sizes,
                    i,                                    
                    levels.d_elems,
                    curLevel,     
                    visited.d_elems,
                    update.d_elems);

                if (util::handleError(hipDeviceSynchronize(), "BFSKernel failed ", __FILE__, __LINE__)) break;

            } else {

                BFSKernel_smart_group_map<VertexId, SizeT, Value, true><<<blockNum, block_size>>>(
                    g.d_row_offsets,
                    g.d_column_indices,
                    workset[selector].d_elems,
                    workset[selector].d_slot_offsets,
                    workset[selector].d_slot_sizes,
                    i,                                    
                    levels.d_elems,
                    curLevel,     
                    visited.d_elems,
                    group_size,
                    group_per_block,
                    update.d_elems);

                if (util::handleError(hipDeviceSynchronize(), "BFSKernel failed ", __FILE__, __LINE__)) break;
            }

            //if (instrument) printf("\t[slot] %d\t%d\t%f\n", i, partialWorksetSize, gpu_timer.elapsedMillis());
        }

        int blockNum = (g.n % block_size == 0) ? 
            (g.n / block_size) :
            (g.n / block_size + 1);
        if (blockNum > 65535) blockNum = 65535;

        // generate the next workset according to update[]
        BFSKernel_smart_gen_workset<<<blockNum, block_size>>> (
            g.n,
            g.d_row_offsets,
            g.d_column_indices,
            visited.d_elems,
            update.d_elems,
            outdegreesLog.d_elems,
            workset[selector ^ 1].d_elems,
            workset[selector ^ 1].d_slot_offsets,
            workset[selector ^ 1].d_slot_sizes);
            
        if (util::handleError(hipDeviceSynchronize(), "BFSKernel failed ", __FILE__, __LINE__)) break;


        // get the new workset size
        worksetSize = workset[selector ^ 1].sum_slot_size();

        
        gpu_timer.stop();
        float level_millis = gpu_timer.elapsedMillis();
        total_millis += level_millis;

        if (instrument) printf("%d\t%d\t%f\n", curLevel, lastWorksetSize, level_millis);


        curLevel += 1;

        // swap the queue
        selector = selector ^ 1;
    }
    
    printf("GPU hashed bfs terminates\n");
    float billion_edges_per_second = (float)g.m / total_millis / 1000000.0;
    printf("Time(s):\t%f\nSpeed(BE/s):\t%f\n", total_millis / 1000.0, billion_edges_per_second);
    //printf("Accumulated Blocks: \t%d\n", accumulatedBlocks);

    levels.print_log();

    levels.del();
    visited.del();
    workset[0].del();
    workset[1].del();
    
}


} // BFS
} // Morgen