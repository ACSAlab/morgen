#include "hip/hip_runtime.h"
/*
 *   The breadth-first search algorithm
 *
 *   Copyright (C) 2013-2014 by
 *   Cheng Yichao        onesuperclark@gmail.com
 *
 *   This program is free software; you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation; either version 2 of the License, or
 *   (at your option) any later version.
 *
 *   This program is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 */


#pragma once
 

#include <morgen/utils/timing.cuh>
#include <morgen/utils/list.cuh>
#include <morgen/utils/var.cuh>


#include <hip/hip_runtime_api.h>

#define INF -1



namespace morgen {

namespace bfs {

/**
 * each thread wakeup and check if activated[tid] == 1
 * using update[] to mark unvisited vertices in this round
 */
template<typename VertexId, typename SizeT, typename Value>
__global__ void
BFSKernel_expand(SizeT     max_size,
		         SizeT     *row_offsets,
                 VertexId  *column_indices,
                 int       *activated,
                 Value     *levels,
                 int       *visited,
                 int       *update)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;


	if (tid < max_size) {

		if (activated[tid] == 1) {

			activated[tid] = 0;     // wakeup only once
			SizeT outEdgeFirst = row_offsets[tid];
			SizeT outEdgeLast = row_offsets[tid+1];

			for (SizeT edge = outEdgeFirst; edge < outEdgeLast; edge++) {

				VertexId inNode = column_indices[edge];
				if (visited[inNode] == 0) {
					levels[inNode] = levels[tid] + 1;
					update[inNode] = 1;
				}
			}
		}
	}
}


/**
 * use update[] to mask activated[]
 */
template<typename VertexId, typename SizeT>
__global__ void
BFSKernel_update(SizeT     max_size,
		         SizeT     *row_offsets,
                 VertexId  *column_indices,
                 int       *activated,
                 int       *visited,
                 int       *update,
                 int       *terminate)
{
	int tid =  blockIdx.x * blockDim.x + threadIdx.x;


	if (tid < max_size) {

		if (update[tid] == 1) {

			activated[tid] = 1;     
			update[tid] = 0;     // clear after activating
			visited[tid] = 1;   
			// as long as one thread try to set it false
			// the while loop will not be terminated 
			*terminate = 0; 
		}
	}
}


template<typename VertexId, typename SizeT, typename Value>
void BFSGraph_gpu_bitmask(const graph::CsrGraph<VertexId, SizeT, Value> &g, VertexId source)
{

	// use a list to represent bitmask
    util::List<int, SizeT> activated(g.n);
    util::List<int, SizeT> update(g.n);
    activated.all_to(0);
    update.all_to(0);

    // Initalize auxiliary list
    util::List<Value, SizeT> levels(g.n);
    levels.all_to(INF);

    // visitation
    util::List<int, SizeT> visited(g.n);
    visited.all_to(0);

    // set up a flag, initially set
    util::Var<int> terminate;
    terminate.set(0);

	// traverse from source node
    activated.set(source, 1);
    levels.set(source, 0);
    visited.set(source, 1);
	Value curLevel = 0;


	// kernel configuration
	int blockSize = 256;
	// spawn as many threads as the vertices in the graph
	int blockNum = (g.n % blockSize == 0 ? 
		g.n / blockSize :
		g.n / blockSize + 1);

	printf("gpu bitmasked bfs starts\n");	
	printf("level\t"
		   "time\n");

	float total_milllis = 0.0;

	// loop as long as the flag is set
	while (terminate.getVal() == 0) {

		// set true at first, if no vertex has been expanded
		// the while loop will be terminated
		terminate.set(1);

		// kick off timer first
		util::GpuTimer gpu_timer;
		gpu_timer.start();

		BFSKernel_expand<<<blockNum, blockSize>>>(g.n,
				                                  g.d_row_offsets,
				            	                  g.d_column_indices,
				                                  activated.d_elems,
				                                  levels.d_elems,             
				                                  visited.d_elems,
				                                  update.d_elems);
		if (util::handleError(hipDeviceSynchronize(), "BFSKernel_expand failed ", __FILE__, __LINE__)) break;

		BFSKernel_update<<<blockNum, blockSize>>>(g.n,
											      g.d_row_offsets,
				            	                  g.d_column_indices,
				                                  activated.d_elems,
				                                  visited.d_elems,
				                                  update.d_elems,     
				                                  terminate.d_elem);
		if (util::handleError(hipDeviceSynchronize(), "BFSKernel_update failed ", __FILE__, __LINE__)) break;


		 // timer end
		 gpu_timer.stop();

		 printf("%d\t%f\n", curLevel, gpu_timer.elapsedMillis());

		 total_milllis += gpu_timer.elapsedMillis();

		 curLevel += 1;

	}
    
    printf("gpu bitmasked bfs terminates\n");
	
    float billion_edges_per_second = (float)g.m / total_milllis / 1000000.0;
    printf("time(s): %f   speed(BE/s): %f\n", total_milllis / 1000.0, billion_edges_per_second);


    levels.print_log();

    levels.del();
    visited.del();
    activated.del();
	update.del();
	terminate.del();
	
}


} // BFS
} // Morgen