#include "hip/hip_runtime.h"

/*
 *   The breadth-first search algorithm
 *
 *   Copyright (C) 2013-2014 by
 *   Cheng Yichao        onesuperclark@gmail.com
 *
 *   This program is free software; you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation; either version 2 of the License, or
 *   (at your option) any later version.
 *
 *   This program is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 */


#pragma once

#include <morgen/utils/macros.cuh>
#include <morgen/utils/timing.cuh>
#include <morgen/utils/list.cuh>
#include <morgen/utils/log.cuh>
#include <morgen/utils/metrics.cuh>
#include <morgen/workset/hash.cuh>

#include <hip/hip_runtime_api.h>


namespace morgen {

namespace bfs {


/**
 * This is a fixed thread-mapping kernel for hashe-based workset
 * The workset of current level is processed in one kernal launch
 */
template<typename VertexId, 
         typename SizeT,
         typename Value>
__global__ void
BFSKernel_topo_thread_map(
  SizeT     *row_offsets,
  VertexId  *column_indices,
  VertexId  *workset_from,
  SizeT     *slot_offsets_from,
  SizeT     *slot_sizes_from,
  int       slot_id_from,
  Value     *levels,
  Value     curLevel,
  int       *update)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;


    if (tid < slot_sizes_from[slot_id_from]) {

        VertexId outNode = workset_from[slot_offsets_from[slot_id_from] + tid];
        SizeT outEdgeFirst = row_offsets[outNode];
        SizeT outEdgeLast = row_offsets[outNode+1];


        for (SizeT e = outEdgeFirst; e < outEdgeLast; e++) {
            VertexId inNode = column_indices[e];

            if (levels[inNode] == MORGEN_INF) { 
                levels[inNode] = curLevel + 1;
                update[inNode] = 1;
            }
            

       }
    }   
    
    
}




template<typename VertexId, 
         typename SizeT, 
         typename Value>
__global__ void
BFSKernel_topo_group_map(
  SizeT     *row_offsets,
  VertexId  *column_indices,
  VertexId  *workset_from,
  SizeT     *slot_offsets_from,
  SizeT     *slot_sizes_from,
  int       slot_id_from,
  Value     *levels,
  Value     curLevel,
  int       *update,
  int       group_size,
  float     group_per_block)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int group_offset = tid % group_size;
    int group_id     = tid / group_size;



    // group_per_block * gridDim.x = how many groups of threads are spawned 
    for (int g = group_id; g < slot_sizes_from[slot_id_from]; g += group_per_block * gridDim.x) {

        VertexId outNode = workset_from[slot_offsets_from[slot_id_from] + g];
        SizeT edgeFirst = row_offsets[outNode];
        SizeT edgeLast = row_offsets[outNode+1];

        // serial expansion
        for (SizeT edge = edgeFirst + group_offset; edge < edgeLast; edge += group_size) 
        {

            VertexId inNode = column_indices[edge];

            if (levels[inNode] == MORGEN_INF) {
                levels[inNode] = curLevel + 1;
                update[inNode] = 1;
            }

        } // edge loop

    }
    
}



/**
 * use update[] to mask activated[]
 */
template<typename VertexId, typename SizeT>
__global__ void
BFSKernel_topo_gen_workset(
    SizeT     max_size,
    SizeT     *row_offsets,
    VertexId  *column_indices,
    int       *update,
    VertexId  *workset_to,
    SizeT     *slot_offsets_to,
    VertexId  *slot_sizes_to,
    int       *outdegrees)
{
    int tid =  blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < max_size) {

        if (update[tid] == 1) {

            update[tid] = 0;     // clear after activating
            int hash = outdegrees[tid];
            if (hash >= 0) {
                SizeT pos= atomicAdd( (SizeT*) &(slot_sizes_to[hash]), 1 );
                workset_to[slot_offsets_to[hash] + pos] = tid;
            }
        }
    }
}





template<typename VertexId, typename SizeT, typename Value>
void BFSGraph_gpu_topo(
    const graph::CsrGraph<VertexId, SizeT, Value> &g, 
    VertexId source, 
    const util::Stats<VertexId, SizeT, Value> &stats,
    bool instrument,
    int block_size,
    bool get_metrics,
    int  static_group_size,
    int threshold,
    int alpha = 100)
{


    // To make better use of the workset, we create two.
    // Instead of creating a new one everytime in each BFS level,
    // we just expand vertices from one to another
    workset::Hash<VertexId, SizeT, Value>  workset(stats, alpha);


    // create a outdegree table first
    // outdegree:     0  (0,1]  (1, 2]  (2, 4]   (4, 8]   (8, 16]
    // altered       -1   0      1       2       3        4       
    util::List<Value, SizeT> outdegreesLog(g.n);
    for (SizeT i = 0; i < g.n; i++) {
        SizeT outDegree = g.row_offsets[i+1] - g.row_offsets[i];

        int slot_should_go;     

        /*   
        if (outDegree == 0) slot_should_go = -1;
        else if (outDegree > 0 && outDegree <= 1) slot_should_go = 0;
        else if (outDegree > 1 && outDegree <= 2) slot_should_go = 1;
        else if (outDegree > 2 && outDegree <= 4) slot_should_go = 2;
        else if (outDegree > 4 && outDegree <= 8) slot_should_go = 3;
        else if (outDegree > 8 && outDegree <= 16) slot_should_go = 4;
        else if (outDegree > 16 && outDegree <= 32) slot_should_go = 5;
        else slot_should_go = 5;
        */

        slot_should_go = util::getLogOf(outDegree);
        // the slot is set to empty, mov it to the next slot
        while (workset.slot_size_max[slot_should_go] == 0) { 
            slot_should_go += 1;
        }

        outdegreesLog.elems[i] = slot_should_go;
        
        //outdegreesLog.elems[i] = util::getLogOf(outDegree);
    }
    outdegreesLog.transfer();



    // Initalize auxiliary list
    util::List<Value, SizeT> levels(g.n);
    levels.all_to((Value) MORGEN_INF);


    // traverse from source node
    workset.insert(outdegreesLog.elems[source], source);   
    levels.set(source, 0);
    util::List<int, SizeT> update(g.n);
    update.all_to(0);


    SizeT worksetSize = 1;
    Value curLevel = 0;
    SizeT edge_frontier_size;

    // kernel configuration
    int blockNum;
    printf("GPU topology-aware bfs starts... \n");  

    if (instrument) printf("level\tslot_size\tfrontier_size\tratio\ttime\n");

    float total_millis = 0.0;
    float level_millis = 0.0;
    float expand_millis = 0.0;
    float compact_millis = 0.0;

    util::Metrics<VertexId, SizeT, Value> metric;

    // kick off timer first
    util::GpuTimer gpu_timer;
    util::GpuTimer expand_timer;
    util::GpuTimer compact_timer;


    gpu_timer.start();

    while (worksetSize > 0) {


        if (instrument) level_millis = 0;

        for (int i = 0; i < workset.slot_num; i++) {

            int partialWorksetSize = workset.slot_sizes[i];
            if (partialWorksetSize== 0) continue;

            // decide which mapping strategy to be used according to i
            int group_size = 0;
            switch (i) {
                case 0: group_size = 1; break;
                case 1: group_size = 2; break;
                case 2: group_size = 4; break;
                case 3: group_size = 8; break;
                case 4: group_size = 16; break;
                case 5: group_size = 32; break;
                default: group_size = 32; 
            }

            while (group_size * partialWorksetSize < threshold) {
                if (group_size == 32) break;
                group_size *= 2;
            }

            if (static_group_size != 0) group_size = static_group_size;

            if (instrument) {
                workset.transfer_back();
                metric.count(workset.elems + workset.slot_offsets[i], partialWorksetSize, g, group_size);
                edge_frontier_size = 0;
                for (int j = 0; j < workset.slot_sizes[i]; j++) {
                    VertexId v = workset.elems[workset.slot_offsets[i]+j];
                    SizeT start = g.row_offsets[v];
                    SizeT end = g.row_offsets[v+1];
                    edge_frontier_size += (end - start);
                }
                expand_timer.start();
            }


            float group_per_block = (float)block_size / group_size;

            blockNum = MORGEN_BLOCK_NUM_SAFE(partialWorksetSize * group_size, block_size);


            if (group_size == 1) {
                BFSKernel_topo_thread_map<VertexId, SizeT, Value><<<blockNum, block_size>>>(
                    g.d_row_offsets,
                    g.d_column_indices,
                    workset.d_elems,
                    workset.d_slot_offsets,
                    workset.d_slot_sizes,
                    i,                                    
                    levels.d_elems,
                    curLevel,     
                    update.d_elems);
            } else {
                BFSKernel_topo_group_map<VertexId, SizeT, Value><<<blockNum, block_size>>>(
                    g.d_row_offsets,
                    g.d_column_indices,
                    workset.d_elems,
                    workset.d_slot_offsets,
                    workset.d_slot_sizes,
                    i,                                    
                    levels.d_elems,
                    curLevel,     
                    update.d_elems,
                    group_size,
                    group_per_block);
            }

            if (util::handleError(hipDeviceSynchronize(), "BFSKernel failed ", __FILE__, __LINE__)) break;

            if (instrument) { 
                expand_timer.stop(); 
                expand_millis +=  expand_timer.elapsedMillis();
                level_millis += expand_timer.elapsedMillis();
            }
            //level_millis += gpu_timer.elapsedMillis();
            if (instrument) printf("\t[slot] %d\t%d\t%d\t%d\t%f\t%f\n", i, group_size, partialWorksetSize, edge_frontier_size, expand_timer.elapsedMillis());
        }


        if (instrument) compact_timer.start();

        // clear the workset first
        workset.clear_slot_sizes();

        blockNum = (g.n % block_size == 0) ? 
            (g.n / block_size) :
            (g.n / block_size + 1);
        if (blockNum > 65535) blockNum = 65535;

        // generate the next workset according to update[]
        BFSKernel_topo_gen_workset<<<blockNum, block_size>>> (
            g.n,
            g.d_row_offsets,
            g.d_column_indices,
            update.d_elems,
            workset.d_elems,
            workset.d_slot_offsets,
            workset.d_slot_sizes,
            outdegreesLog.d_elems);

        if (util::handleError(hipDeviceSynchronize(), "BFSKernel failed ", __FILE__, __LINE__)) break;

        // get the size of workset

        if (instrument) {
            compact_timer.stop();
            compact_millis +=  compact_timer.elapsedMillis();
            printf("%d\t%d\t%f\t%f\n", curLevel, worksetSize, level_millis, compact_timer.elapsedMillis());
        }

        worksetSize = workset.sum_slot_size();

        //total_millis += level_millis;
        curLevel += 1;
    }

    gpu_timer.stop();
    total_millis = gpu_timer.elapsedMillis();


    printf("GPU topo bfs terminates\n");
    float billion_edges_per_second = (float)g.m / total_millis / 1000000.0;
    printf("Time(s):\t%f\nSpeed(BE/s):\t%f\n", total_millis / 1000.0, billion_edges_per_second);
    //printf("Accumulated Blocks: \t%d\n", accumulatedBlocks);


    if (instrument) printf("Expand: \t%f\t%f\n", expand_millis / 1000.0, compact_millis / 1000.0);
    if (instrument) metric.display();


    levels.print_log();

    levels.del();
    update.del();
    outdegreesLog.del();
    workset.del();
    
}


} // BFS
} // Morgen