#include "hip/hip_runtime.h"
/*
 *   The breadth-first search algorithm
 *
 *   Copyright (C) 2013-2014 by
 *   Cheng Yichao        onesuperclark@gmail.com
 *
 *   This program is free software; you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation; either version 2 of the License, or
 *   (at your option) any later version.
 *
 *   This program is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 */


#pragma once
 
#include <morgen/utils/macros.cuh>
#include <morgen/utils/timing.cuh>
#include <morgen/utils/list.cuh>
#include <morgen/utils/var.cuh>


#include <morgen/graph/coo_edge_tuple.cuh>


#include <hip/hip_runtime_api.h>




namespace morgen {

namespace bfs {

/**
 * each thread wakeup and check if activated[tid] == 1
 * using update[] to mark unvisited vertices in this round
 */
template<typename VertexId, typename SizeT, typename Value>
__global__ void
BFSKernel_expand_coo(
  SizeT            max_size,
  morgen::graph::CooEdgeTuple<VertexId>     *elems,
  int              *activated,
  Value            *levels,
  Value            curLevel,
  int              *visited,
  int              *update)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < max_size) {

        VertexId src = elems[tid].row;

        if (activated[src] == 1) {
            activated[src] = 0;     // wakeup only once
            VertexId dest = elems[tid].col;
            if (visited[dest] == 0) {
                levels[dest] = curLevel + 1;
                update[dest] = 1;
            }
        }
    }
}



/**
 * use update[] to mask activated[]
 */
template<typename SizeT>
__global__ void
BFSKernel_update_coo(
    SizeT     max_size,
    int       *activated,
    int       *visited,
    int       *update,
    int       *terminate)
{
    int tid =  blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < max_size) {
        if (update[tid] == 1) {
            activated[tid] = 1;     
            update[tid] = 0;     // clear after activating
            visited[tid] = 1;   
            *terminate = 0; 
        }
    }
}


template<typename VertexId, typename SizeT, typename Value>
void BFSGraph_gpu_bitmask_coo(
    const graph::CooGraph<VertexId, SizeT, Value> &g,
    VertexId source,
    int block_size,
    int instrument)
{

    // use a list to represent bitmask
    util::List<int, SizeT> activated(g.n);
    util::List<int, SizeT> update(g.n);
    util::List<Value, SizeT> levels(g.n);
    util::List<int, SizeT> visited(g.n);
    util::Var<int> terminate;

    activated.all_to(0);
    update.all_to(0);
    levels.all_to((Value) MORGEN_INF);
    visited.all_to(0);
    terminate.set(0);

    // traverse from source node
    activated.set(source, 1);
    levels.set(source, 0);
    visited.set(source, 1);
    Value curLevel = 0;



    printf("GPU bitmasked bfs starts... \n");   
    if (instrument) printf("level\ttime\n");

    float total_milllis = 0.0;


    // loop as long as the flag is set
    while (terminate.getVal() == 0) {

        // set true at first, if no vertex has been expanded
        // the while loop will be terminated
        terminate.set(1);

        // kick off timer first
        util::GpuTimer gpu_timer;
        gpu_timer.start();


        int blockNum = MORGEN_BLOCK_NUM_SAFE(g.m, block_size);
        BFSKernel_expand_coo<<<blockNum, block_size>>>(
            g.m,
            g.d_elems,
            activated.d_elems,
            levels.d_elems,
            curLevel,             
            visited.d_elems,
            update.d_elems);

        if (util::handleError(hipDeviceSynchronize(), "BFSKernel_expand failed ", __FILE__, __LINE__)) break;



        blockNum = MORGEN_BLOCK_NUM_SAFE(g.n, block_size);
        BFSKernel_update_coo<<<blockNum, block_size>>>(
            g.n,
            activated.d_elems,
            visited.d_elems,
            update.d_elems,     
            terminate.d_elem);
        
        if (util::handleError(hipDeviceSynchronize(), "BFSKernel_update failed ", __FILE__, __LINE__)) break;


         // timer end
         gpu_timer.stop();

         if (instrument) printf("%d\t%f\n", curLevel, gpu_timer.elapsedMillis());
         total_milllis += gpu_timer.elapsedMillis();
         curLevel += 1;

    }
    
    printf("GPU bitmasked bfs terminates\n");
    float billion_edges_per_second = (float)g.m / total_milllis / 1000000.0;
    printf("Time(s):\t%f\nSpeed(BE/s):\t%f\n", total_milllis / 1000.0, billion_edges_per_second);


    levels.print_log();

    levels.del();
    visited.del();
    activated.del();
    update.del();
    terminate.del();
    
}


} // BFS
} // Morgen