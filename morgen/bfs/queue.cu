#include "hip/hip_runtime.h"
/*
 *   The breadth-first search algorithm
 *
 *   Copyright (C) 2013-2014 by
 *   Cheng Yichao        onesuperclark@gmail.com
 *
 *   This program is free software; you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation; either version 2 of the License, or
 *   (at your option) any later version.
 *
 *   This program is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 */


#pragma once

#include <morgen/utils/timing.cuh>
#include <morgen/utils/list.cuh>
#include <morgen/workset/queue.cuh>

#include <hip/hip_runtime_api.h>


#define INF    -1
#define WARP_S 32

namespace morgen {

namespace bfs {

template<typename VertexId, typename SizeT, typename Value>
__global__ void
BFSKernel(SizeT     *row_offsets,
          VertexId  *column_indices,
          VertexId  *worksetFrom,
          SizeT     *sizeFrom,
          VertexId  *worksetTo,
          SizeT     *sizeTo,
          Value     *levels,
          Value     curLevel,
          int       *visited)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // writing to an empty buffer
    if (tid == 0) *sizeTo = 0;
    __syncthreads();

    if (tid < *sizeFrom) {
        
        // read the who-am-I info from the workset
        VertexId outNode = worksetFrom[tid];
        levels[outNode] = curLevel;

        SizeT outEdgeFirst = row_offsets[outNode];
        SizeT outEdgeLast = row_offsets[outNode+1];

        // serial expansion
        for (SizeT edge = outEdgeFirst; edge < outEdgeLast; edge++) {

            VertexId inNode = column_indices[edge];

            int old = atomicExch( (int*)&visited[inNode], 1 );

            if (old == 0) { 
                // fine-grained allocation
                SizeT pos= atomicAdd( (SizeT*) &(*sizeTo), 1 );
                worksetTo[pos] = inNode;
            }
        }   

    }
}


template<typename VertexId, typename SizeT, typename Value>
__global__ void
BFSKernel_warp_mapped(SizeT     *row_offsets,
                      VertexId  *column_indices,
                      VertexId  *worksetFrom,
                      SizeT     *sizeFrom,
                      VertexId  *worksetTo,
                      SizeT     *sizeTo,
                      Value     *levels,
                      Value     curLevel,
                      int       *visited)
{



    int tid = blockIdx.x * blockDim.x + threadIdx.x;


    int warp_offset = tid % WARP_S;
    int warp_id     = tid / WARP_S;

    // writing to an empty buffer
    if (tid == 0) *sizeTo = 0;

    __syncthreads();


    // each warp has one variable to record how many neighbor nodes
    // it will process
    // warps per block: thread per block / warp size

    volatile __shared__ SizeT edge_first[8];
    volatile __shared__ SizeT edge_last[8];


    if (warp_id < *sizeFrom) {

        // First thread in a warp do this job
        // read out info from global mem to local mem
        if (warp_offset == 0) {
            VertexId outNode = worksetFrom[warp_id];
            levels[outNode] = curLevel;


            edge_first[warp_id % 8] = row_offsets[outNode];
            edge_last[warp_id % 8] = row_offsets[outNode+1];
        }

        __syncthreads();

        
        // in case the neighbor number > warp size
        for (SizeT edge = edge_first[warp_id % 8] + warp_offset;
             edge < edge_last[warp_id % 8];
             edge += WARP_S)
        {
            
            VertexId inNode = column_indices[edge];

            int old = atomicExch( (int*)&visited[inNode], 1 );

            if (old == 0) { 
                // fine-grained allocation
                SizeT pos= atomicAdd( (SizeT*) &(*sizeTo), 1 );
                worksetTo[pos] = inNode;
            }
        }

    
    }
}



template<typename VertexId, typename SizeT, typename Value>
void BFSGraph_gpu_queue(
    const graph::CsrGraph<VertexId, SizeT, Value> &g,
    VertexId source,
    bool instrument,
    int blockSize,
    bool warp_mapped)
{

    // To make better use of the workset, we create two.
    // Instead of creating a new one everytime in each BFS level,
    // we just expand vertices from one to another
    workset::Queue<VertexId, SizeT> workset1(g.n);
    workset::Queue<VertexId, SizeT> workset2(g.n);


    // Initalize auxiliary list
    util::List<Value, SizeT> levels(g.n);
    levels.all_to(INF);


    // visitation list: 0 for unvisited
    util::List<int, SizeT> visited(g.n);
    visited.all_to(0);


    // traverse from source node
    workset1.append(source);   
    levels.set(source, 0);
    visited.set(source, 1);
    SizeT worksetSize = 1;
    SizeT lastWorksetSize = 0;
    Value curLevel = 0;

    // kernel configuration
    int blockNum = 16;
    

    printf("gpu queued bfs starts... \n");  

    if (instrument) printf("level\tfrontier_size\tblock_num\ttime\n");


    float total_milllis = 0.0;

    // how many threads are mapped to a work element
    int mapping_factor = (warp_mapped) ? WARP_S : 1; 

    while (worksetSize > 0) {

        lastWorksetSize = worksetSize;


        // spawn minimal software blocks to cover the workset
        blockNum = (worksetSize * mapping_factor % blockSize == 0 ? 
            worksetSize * mapping_factor / blockSize :
            worksetSize * mapping_factor / blockSize + 1);


        // kick off timer first
        util::GpuTimer gpu_timer;
        gpu_timer.start();

        if (curLevel % 2 == 0) 
        {


            if (warp_mapped) {
                BFSKernel_warp_mapped<<<blockNum, blockSize>>>(g.d_row_offsets,
                                                               g.d_column_indices,
                                                               workset1.d_elems,
                                                               workset1.d_sizep,
                                                               workset2.d_elems,
                                                               workset2.d_sizep,
                                                               levels.d_elems,
                                                               curLevel,     
                                                               visited.d_elems);

            } else {
                // call kernel with device pointers
                BFSKernel<<<blockNum, blockSize>>>(g.d_row_offsets,
                                                   g.d_column_indices,
                                                   workset1.d_elems,
                                                   workset1.d_sizep,
                                                   workset2.d_elems,
                                                   workset2.d_sizep,
                                                   levels.d_elems,
                                                   curLevel,     
                                                   visited.d_elems);
            }


            if (util::handleError(hipDeviceSynchronize(), "BFSKernel failed ", __FILE__, __LINE__)) break;

            worksetSize = workset2.size();

         } else {

            if (warp_mapped) {
                BFSKernel_warp_mapped<<<blockNum , blockSize>>>(g.d_row_offsets,
                                                                g.d_column_indices,
                                                                workset2.d_elems,
                                                                workset2.d_sizep,
                                                                workset1.d_elems,
                                                                workset1.d_sizep,
                                                                levels.d_elems,
                                                                curLevel,     
                                                                visited.d_elems);

            } else {
                BFSKernel<<<blockNum, blockSize>>>(g.d_row_offsets,
                                                   g.d_column_indices,
                                                   workset2.d_elems,
                                                   workset2.d_sizep,
                                                   workset1.d_elems,
                                                   workset1.d_sizep,
                                                   levels.d_elems,
                                                   curLevel,
                                                   visited.d_elems);
            }



            if (util::handleError(hipDeviceSynchronize(), "BFSKernel failed ", __FILE__, __LINE__)) break;

            
            worksetSize = workset1.size();
         }

         // timer end
         gpu_timer.stop();

         if (instrument) printf("%d\t%d\t%d\t%f\n", curLevel, lastWorksetSize, blockNum, gpu_timer.elapsedMillis());
         
         total_milllis += gpu_timer.elapsedMillis();
         curLevel += 1;

    }
    
    printf("gpu queued bfs terminates\n");  
    float billion_edges_per_second = (float)g.m / total_milllis / 1000000.0;
    printf("time(s): %f   speed(BE/s): %f\n", total_milllis / 1000.0, billion_edges_per_second);


    levels.print_log();

    levels.del();
    visited.del();
    workset1.del();
    workset2.del();
    
}


} // BFS

} // Morgen