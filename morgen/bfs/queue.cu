#include "hip/hip_runtime.h"
/*
 *   The breadth-first search algorithm
 *
 *   Copyright (C) 2013-2014 by
 *   Cheng Yichao        onesuperclark@gmail.com
 *
 *   This program is free software; you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation; either version 2 of the License, or
 *   (at your option) any later version.
 *
 *   This program is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 */


#pragma once


#include <morgen/utils/macros.cuh>
#include <morgen/utils/timing.cuh>
#include <morgen/utils/list.cuh>
#include <morgen/utils/metrics.cuh>
#include <morgen/workset/queue.cuh>
#include <hip/hip_runtime_api.h>


namespace morgen {

namespace bfs {


/* texture memory */
//texture<int> tex_row_offsets;
//texture<int> tex_column_indices;


template<typename VertexId,
         typename SizeT, 
         typename Value>
__global__ void
BFSKernel_queue_thread_map(
    SizeT     *row_offsets,
    VertexId  *column_indices,
    VertexId  *worksetFrom,
    SizeT     *sizeFrom,
    Value     *levels,
    Value     curLevel,
    int       *update)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < *sizeFrom) {
        
        // read the who-am-I info from the workset
        VertexId outNode = worksetFrom[tid];

        SizeT outEdgeFirst = row_offsets[outNode];
        //SizeT outEdgeFirst = tex1Dfetch(tex_row_offsets, outNode);

        SizeT outEdgeLast = row_offsets[outNode+1];
        //SizeT outEdgeLast = tex1Dfetch(tex_row_offsets, outNode+1);

        // serial expansion
        for (SizeT edge = outEdgeFirst; edge < outEdgeLast; edge++) {

            VertexId inNode = column_indices[edge];
            //VertexId inNode = tex1Dfetch(tex_column_indices, edge);



            if (levels[inNode] == MORGEN_INF) {
                levels[inNode] = curLevel + 1;
                update[inNode] = 1;
            }

        }   
    }
}




/**
 * Each vertex(u) in worksetFrom is assigned with a group of threads.
 * Then each thead within a group processes one of u's neigbors
 * at a time. All threads process vertices in SIMD manner.
 *
 * Assume GROUP_S = 32
 * If u has a neigbor number more than 32, each thead within a group will 
 * iterate over them stridedly. e.g. thread 1 will process 1st, 33th, 65th... 
 * vertex in the neighbor list, thread 2 will process 2nd, 34th, 66th...
 */
template<typename VertexId, 
         typename SizeT, 
         typename Value>
__global__ void
BFSKernel_queue_group_map(
    SizeT     *row_offsets,
    VertexId  *column_indices,
    VertexId  *worksetFrom,
    SizeT     *sizeFrom,
    Value     *levels,
    Value     curLevel,
    int       group_size,
    float     group_per_block,
    int       *update)
{

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int group_offset = tid % group_size;
    int group_id     = tid / group_size;


    // Since the workset can easily exceed 65536, we just let grouped-threads
    // iterate over a large workset
    for (int g = group_id; g < *sizeFrom; g += group_per_block * gridDim.x) {


        VertexId outNode = worksetFrom[g];
        SizeT edgeFirst = row_offsets[outNode];
        SizeT edgeLast = row_offsets[outNode+1];

        // in case the neighbor number > warp size
        for (SizeT edge = edgeFirst + group_offset; edge < edgeLast; edge += group_size)
        {
            
            VertexId inNode = column_indices[edge];
            //VertexId inNode = tex1Dfetch(tex_column_indices, edge);


            if (levels[inNode] == MORGEN_INF) {
                levels[inNode] = curLevel + 1;
                update[inNode] = 1;
            }

        }
    }
}


/**
 * use update[] to mask activated[]
 */
template<typename VertexId, typename SizeT>
__global__ void
BFSKernel_queue_gen_workset(
    SizeT     max_size,
    SizeT     *row_offsets,
    VertexId  *column_indices,
    int       *update,
    VertexId  *worksetTo,
    SizeT     *sizeTo)
{
    int tid =  blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < max_size) {

        if (update[tid] == 1) {

            update[tid] = 0;     // clear after activating

            SizeT pos = atomicAdd( (SizeT*) &(*sizeTo), 1 );
            worksetTo[pos] = tid;
        }
    }
}



template<typename VertexId, typename SizeT, typename Value>
void BFSGraph_gpu_queue(
    const graph::CsrGraph<VertexId, SizeT, Value> &g,
    VertexId source,
    bool instrument,
    int block_size,
    bool warp_mapped,
    int group_size,
    bool get_metrics)
{

    // To make better use of the workset, we create two.
    // Instead of creating a new one everytime in each BFS level,
    // we just expand vertices from one to another
    workset::Queue<VertexId, SizeT>  workset(g.n);

    // Initalize auxiliary list
    util::List<Value, SizeT> levels(g.n);
    levels.all_to((Value) MORGEN_INF);


    util::List<int, SizeT> update(g.n);
    update.all_to(0);

    // traverse from source node
    workset.init(source);   
    levels.set(source, 0);
    
    SizeT worksetSize = 1;
    //SizeT lastWorksetSize = 0;
    Value curLevel = 0;

    SizeT edge_frontier_size;

    float total_millis = 0.0;
    float expand_millis = 0.0;
    float compact_millis = 0.0;

    if (warp_mapped == false) group_size = 1;
    float group_per_block = (float)block_size / group_size;

    printf("GPU queued bfs starts... \n");  
    if (instrument) printf("level\tfrontier_size\tblock_num\ttime\n");


    util::Metrics<VertexId, SizeT, Value> metric;
    util::Metrics<VertexId, SizeT, Value> level_metric;

    /* 

    bind the graph in texture memory(1D)
    
    if (util::handleError(hipBindTexture(0, tex_column_indices, g.d_column_indices, sizeof(VertexId) * g.m), 
        "CsrGraph: bindTexture(d_column_indices) failed", __FILE__, __LINE__)) exit(1);        

    if (util::handleError(hipBindTexture(0, tex_row_offsets, g.d_row_offsets, sizeof(SizeT) * (g.n + 1)), 
        "CsrGraph: bindTexture(d_row_offsets) failed", __FILE__, __LINE__)) exit(1);
        
    printf("Done texture memory binding.\n");

    */


    util::GpuTimer gpu_timer;
    util::GpuTimer expand_timer;
    util::GpuTimer compact_timer;


    gpu_timer.start();


    while (worksetSize > 0) {

        if (instrument) {
            workset.transfer_back();
            metric.count(workset.elems, workset.size(), g, group_size);
            edge_frontier_size = 0;
            for (int i = 0; i < *workset.sizep; i++) {
                VertexId v = workset.elems[i];
                SizeT start = g.row_offsets[v];
                SizeT end = g.row_offsets[v+1];
                edge_frontier_size += (end - start);
            }
            expand_timer.start();  // start timer
        }


        // spawn minimal(but enough) software blocks to cover the workset
        int blockNum = (worksetSize * group_size % block_size == 0 ? 
            worksetSize * group_size / block_size :
            worksetSize * group_size/ block_size + 1);
        
        // safe belt: grid width has a limit of 65535
        if (blockNum > 65535) blockNum = 65535;

        if (warp_mapped) {

            BFSKernel_queue_group_map<VertexId, SizeT, Value><<<blockNum, block_size>>>(
                g.d_row_offsets,
                g.d_column_indices,
                workset.d_elems,
                workset.d_sizep,
                levels.d_elems,
                curLevel,     
                group_size,
                group_per_block,
                update.d_elems);

        } else { // thread map

            BFSKernel_queue_thread_map<VertexId, SizeT, Value><<<blockNum, block_size>>>(
                g.d_row_offsets,                                        
                g.d_column_indices,
                workset.d_elems,
                workset.d_sizep,
                levels.d_elems,
                curLevel,     
                update.d_elems);

        }
        if (util::handleError(hipDeviceSynchronize(), "BFSKernel failed ", __FILE__, __LINE__)) break;

        if (instrument) {
            expand_timer.stop();
            expand_millis += expand_timer.elapsedMillis();
            compact_timer.start();
        }      

        workset.clear_size();


        blockNum = (g.n % block_size == 0) ? 
            (g.n / block_size) :
            (g.n / block_size + 1);
        if (blockNum > 65535) blockNum = 65535;

  
        // generate the next workset according to update[]
        BFSKernel_queue_gen_workset<<<blockNum, block_size>>> (
            g.n,
            g.d_row_offsets,
            g.d_column_indices,
            update.d_elems,
            workset.d_elems,
            workset.d_sizep);

        if (util::handleError(hipDeviceSynchronize(), "BFSKernel failed ", __FILE__, __LINE__)) break;


        if (instrument) {
            compact_timer.stop();
            compact_millis += compact_timer.elapsedMillis();
            printf("%d\t%d\t%d\t%f\t%f\n", curLevel, worksetSize, edge_frontier_size, expand_timer.elapsedMillis(), compact_timer.elapsedMillis());
        }

        worksetSize = workset.size();

        curLevel += 1;

    } // endwhile


    gpu_timer.stop();
    total_millis = gpu_timer.elapsedMillis();


    printf("GPU queued bfs terminates\n");  
    float billion_edges_per_second = (float)g.m / total_millis / 1000000.0;
    printf("Time(s):\t%f\nSpeed(BE/s):\t%f\n", total_millis / 1000.0, billion_edges_per_second);
    //printf("Accumulated Blocks: \t%d\n", accumulatedBlocks);
    if (instrument) printf("Expand:\t%f\t%f\n", expand_millis / 1000.0, compact_millis / 1000.0);
    if (instrument) metric.display();



    levels.print_log();

    levels.del();
    update.del();
    workset.del();
    
}


} // BFS

} // Morgen