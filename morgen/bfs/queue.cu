#include "hip/hip_runtime.h"
/*
 *   The breadth-first search algorithm
 *
 *   Copyright (C) 2013-2014 by
 *   Cheng Yichao        onesuperclark@gmail.com
 *
 *   This program is free software; you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation; either version 2 of the License, or
 *   (at your option) any later version.
 *
 *   This program is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 */


#pragma once


#include <morgen/utils/macros.cuh>
#include <morgen/utils/timing.cuh>
#include <morgen/utils/list.cuh>
#include <morgen/utils/metrics.cuh>
#include <morgen/workset/queue.cuh>
#include <hip/hip_runtime_api.h>


namespace morgen {

namespace bfs {



/**
 * Each vertex(u) in worksetFrom is assigned with a group of threads.
 * Then each thead within a group processes one of u's neigbors
 * at a time. All threads process vertices in SIMD manner.
 *
 * Assume GROUP_S = 32
 * If u has a neigbor number more than 32, each thead within a group will 
 * iterate over them stridedly. e.g. thread 1 will process 1st, 33th, 65th... 
 * vertex in the neighbor list, thread 2 will process 2nd, 34th, 66th...
 */
template<typename VertexId, 
         typename SizeT, 
         typename Value>
__global__ void
BFSKernel_queue_group_map(
    SizeT     *row_offsets,
    VertexId  *column_indices,
    VertexId  *worksetFrom,
    SizeT     *sizeFrom,
    Value     *levels,
    Value     curLevel,
    int       group_size,
    float     group_per_block,
    int       *update)
{

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int group_offset = tid % group_size;
    int group_id     = tid / group_size;


    // Since the workset can easily exceed 65536, we just let grouped-threads
    // iterate over a large workset
    for (int g = group_id; g < *sizeFrom; g += group_per_block * gridDim.x) {


        VertexId outNode = worksetFrom[g];
        SizeT edgeFirst = row_offsets[outNode];
        SizeT edgeLast = row_offsets[outNode+1];

        // in case the neighbor number > warp size
        for (SizeT edge = edgeFirst + group_offset; edge < edgeLast; edge += group_size)
        {
            
            VertexId inNode = column_indices[edge];

            if (levels[inNode] == MORGEN_INF) {
                levels[inNode] = curLevel + 1;
                update[inNode] = 1;
            }

        }
    }
}


/**
 * use update[] to mask activated[]
 */
template<typename VertexId, typename SizeT>
__global__ void
BFSKernel_queue_gen_workset(
    SizeT     max_size,
    SizeT     *row_offsets,
    VertexId  *column_indices,
    int       *update,
    VertexId  *worksetTo,
    SizeT     *sizeTo)
{
    int tid =  blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < max_size) {

        if (update[tid] == 1) {

            update[tid] = 0;     // clear after activating

            SizeT pos = atomicAdd( (SizeT*) &(*sizeTo), 1 );
            worksetTo[pos] = tid;
        }
    }
}



template<typename VertexId, typename SizeT, typename Value>
void BFSGraph_gpu_queue(
    const graph::CsrGraph<VertexId, SizeT, Value> &g,
    VertexId source,
    bool instrument,
    int block_size,
    int group_size,
    bool get_metrics)
{

    workset::Queue<VertexId, SizeT>  workset(g.n);

    util::List<Value, SizeT> levels(g.n);
    levels.all_to((Value) MORGEN_INF);

    util::List<int, SizeT> update(g.n);
    update.all_to(0);

    workset.init(source);   
    levels.set(source, 0);
    
    SizeT worksetSize = 1;
    Value curLevel = 0;

    SizeT edge_frontier_size;

    float total_millis = 0.0;
    float expand_millis = 0.0;
    float compact_millis = 0.0;

    float group_per_block = (float)block_size / group_size;

    printf("GPU queued bfs starts... \n");  
    if (instrument) printf("level\tfrontier_size\tblock_num\ttime\n");


    util::Metrics<VertexId, SizeT, Value> metric;
    util::Metrics<VertexId, SizeT, Value> level_metric;


    util::GpuTimer gpu_timer;
    util::GpuTimer expand_timer;
    util::GpuTimer compact_timer;


    gpu_timer.start();


    while (worksetSize > 0) {

        if (instrument) {
            workset.transfer_back();
            metric.count(workset.elems, workset.size(), g, group_size);
            edge_frontier_size = 0;
            for (int i = 0; i < *workset.sizep; i++) {
                VertexId v = workset.elems[i];
                SizeT start = g.row_offsets[v];
                SizeT end = g.row_offsets[v+1];
                edge_frontier_size += (end - start);
            }
            expand_timer.start();  // start timer
        }

        int blockNum = MORGEN_BLOCK_NUM_SAFE(worksetSize * group_size, block_size);



        BFSKernel_queue_group_map<VertexId, SizeT, Value><<<blockNum, block_size>>>(
            g.d_row_offsets,
            g.d_column_indices,
            workset.d_elems,
            workset.d_sizep,
            levels.d_elems,
            curLevel,     
            group_size,
            group_per_block,
            update.d_elems);

        if (util::handleError(hipDeviceSynchronize(), "BFSKernel failed ", __FILE__, __LINE__)) break;

        if (instrument) {
            expand_timer.stop();
            expand_millis += expand_timer.elapsedMillis();
            compact_timer.start();
        }      

        workset.clear_size();

        blockNum = MORGEN_BLOCK_NUM_SAFE(g.n, block_size);
  
        // generate the next workset according to update[]
        BFSKernel_queue_gen_workset<<<blockNum, block_size>>> (
            g.n,
            g.d_row_offsets,
            g.d_column_indices,
            update.d_elems,
            workset.d_elems,
            workset.d_sizep);

        if (util::handleError(hipDeviceSynchronize(), "BFSKernel failed ", __FILE__, __LINE__)) break;

        if (instrument) {
            compact_timer.stop();
            compact_millis += compact_timer.elapsedMillis();
            printf("%d\t%d\t%d\t%f\t%f\n", curLevel, worksetSize, edge_frontier_size, expand_timer.elapsedMillis(), compact_timer.elapsedMillis());
        }

        worksetSize = workset.size();

        curLevel += 1;

    } // endwhile


    gpu_timer.stop();
    total_millis = gpu_timer.elapsedMillis();

    printf("GPU queued bfs terminates\n");  
    float billion_edges_per_second = (float)g.m / total_millis / 1000000.0;
    printf("Time(s):\t%f\nSpeed(BE/s):\t%f\n", total_millis / 1000.0, billion_edges_per_second);
    //printf("Accumulated Blocks: \t%d\n", accumulatedBlocks);
    if (instrument) printf("Expand:\t%f\t%f\n", expand_millis / 1000.0, compact_millis / 1000.0);
    if (instrument) metric.display();


    levels.print_log();

    levels.del();
    update.del();
    workset.del();
    
}


} // BFS

} // Morgen