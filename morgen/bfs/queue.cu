#include "hip/hip_runtime.h"
/*
 *   The breadth-first search algorithm
 *
 *   Copyright (C) 2013-2014 by
 *   Cheng Yichao        onesuperclark@gmail.com
 *
 *   This program is free software; you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation; either version 2 of the License, or
 *   (at your option) any later version.
 *
 *   This program is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 */


#pragma once


#include <morgen/utils/macros.cuh>
#include <morgen/utils/timing.cuh>
#include <morgen/utils/list.cuh>
#include <morgen/workset/queue.cuh>
#include <hip/hip_runtime_api.h>


namespace morgen {

namespace bfs {


/* texture memory */
texture<int> tex_row_offsets;
texture<int> tex_column_indices;


template<typename VertexId,
         typename SizeT, 
         typename Value,
         bool ORDERED>
__global__ void
BFSKernel_queue_thread_map(
    SizeT     *row_offsets,
    VertexId  *column_indices,
    VertexId  *worksetFrom,
    SizeT     *sizeFrom,
    VertexId  *worksetTo,
    SizeT     *sizeTo,
    Value     *levels,
    Value     curLevel,
    int       *visited)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // writing to an empty buffer
    if (tid == 0) *sizeTo = 0;
    __syncthreads();

    if (tid < *sizeFrom) {
        
        // read the who-am-I info from the workset
        VertexId outNode = worksetFrom[tid];

        //SizeT outEdgeFirst = row_offsets[outNode];
        SizeT outEdgeFirst = tex1Dfetch(tex_row_offsets, outNode);

        //SizeT outEdgeLast = row_offsets[outNode+1];
        SizeT outEdgeLast = tex1Dfetch(tex_row_offsets, outNode+1);

        // serial expansion
        for (SizeT edge = outEdgeFirst; edge < outEdgeLast; edge++) {

            //VertexId inNode = column_indices[edge];
            VertexId inNode = tex1Dfetch(tex_column_indices, edge);

            Value level = curLevel + 1;

            if (ORDERED) {
                int old = atomicExch( (int*)&visited[inNode], 1 );
                if (old == 0) { 
                    levels[inNode] = level;
                    SizeT pos= atomicAdd( (SizeT*) &(*sizeTo), 1 );
                    worksetTo[pos] = inNode;
                }
            } else {
                if (levels[inNode] > level) {
                    levels[inNode] = level;     // relax
                    SizeT pos= atomicAdd( (SizeT*) &(*sizeTo), 1 );
                    worksetTo[pos] = inNode;
                }
            }
        }   
    }
}




/**
 * Each vertex(u) in worksetFrom is assigned with a group of threads.
 * Then each thead within a group processes one of u's neigbors
 * at a time. All threads process vertices in SIMD manner.
 *
 * Assume GROUP_S = 32
 * If u has a neigbor number more than 32, each thead within a group will 
 * iterate over them stridedly. e.g. thread 1 will process 1st, 33th, 65th... 
 * vertex in the neighbor list, thread 2 will process 2nd, 34th, 66th...
 */
template<typename VertexId, 
         typename SizeT, 
         typename Value,
         bool ORDERED>
__global__ void
BFSKernel_queue_group_map(
    SizeT     *row_offsets,
    VertexId  *column_indices,
    VertexId  *worksetFrom,
    SizeT     *sizeFrom,
    VertexId  *worksetTo,
    SizeT     *sizeTo,
    Value     *levels,
    Value     curLevel,
    int       *visited,
    int       group_size,
    int       group_per_block)
{

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int group_offset = tid % group_size;
    int group_id     = tid / group_size;

    // writing to an empty buffer
    if (tid == 0) *sizeTo = 0;

    __syncthreads();


    // Each group has use a variable to record the total work
    // amount(neigbors) that belongs to that group
    // groups/block = thread per block / group size
    // The size is allocated dynamically
    volatile __shared__ SizeT edge_first[256];
    volatile __shared__ SizeT edge_last[256];


    // Since the workset can easily exceed 65536, we just let grouped-threads
    // iterate over a large workset
    for (int g = group_id; g < *sizeFrom; g += group_per_block * gridDim.x) {

        //if (g % 1024 == 0 && group_offset == 0)
        //    printf("I am group %d, size: %d, my next: %d\n", g, *sizeFrom, g+group_per_block * gridDim.x);


        // First thread in the group do this job read out info 
        // from global mem to local mem
        if (group_offset == 0) {

            VertexId outNode = worksetFrom[g];
            edge_first[group_id % group_per_block] = row_offsets[outNode];
            edge_last[group_id % group_per_block] = row_offsets[outNode+1];
        }

        __syncthreads();
    
        // in case the neighbor number > warp size
        for (SizeT edge = edge_first[group_id % group_per_block] + group_offset;
             edge < edge_last[group_id % group_per_block];
             edge += group_size)
        {
            
            VertexId inNode = column_indices[edge];

            Value level = curLevel + 1;

            if (ORDERED) {
                int old = atomicExch( (int*)&visited[inNode], 1 );
                if (old == 0) { 
                    levels[inNode] = level;
                    SizeT pos= atomicAdd( (SizeT*) &(*sizeTo), 1 );
                    worksetTo[pos] = inNode;
                } 
            } else {
                if (levels[inNode] > level) {
                    levels[inNode] = level;
                    SizeT pos= atomicAdd( (SizeT*) &(*sizeTo), 1 );
                    worksetTo[pos] = inNode;
                }
            } // ordered
        }
    }
}



template<typename VertexId, typename SizeT, typename Value>
void BFSGraph_gpu_queue(
    const graph::CsrGraph<VertexId, SizeT, Value> &g,
    VertexId source,
    bool instrument,
    int block_size,
    bool warp_mapped,
    int group_size,
    bool unordered)
{

    // To make better use of the workset, we create two.
    // Instead of creating a new one everytime in each BFS level,
    // we just expand vertices from one to another
    workset::Queue<VertexId, SizeT>  workset[] = {
        workset::Queue<VertexId, SizeT>(g.n),
        workset::Queue<VertexId, SizeT>(g.n),
    };

    // use to select between two worksets
    // src:  workset[selector]
    // dest: workset[selector ^ 1]
    int selector = 0;

    // Initalize auxiliary list
    util::List<Value, SizeT> levels(g.n);
    levels.all_to((Value) MORGEN_INF);

    // visitation list: 0 for unvisited
    util::List<int, SizeT> visited(g.n);
    visited.all_to(0);

    // traverse from source node
    workset[0].init(source);   
    levels.set(source, 0);
    visited.set(source, 1);
    
    SizeT worksetSize = 1;
    SizeT lastWorksetSize = 0;
    Value curLevel = 0;
    float total_milllis = 0.0;
    int accumulatedBlocks = 0;

    // kernel configuration
    int blockNum;
    int mapping_factor = (warp_mapped) ? group_size : 1; 
    int group_per_block = block_size / group_size;

    printf("GPU queued bfs starts... \n");  
    if (instrument) printf("level\tfrontier_size\tblock_num\ttime\n");


    /*
    * bind the graph in texture memory(1D)
    */
    if (util::handleError(hipBindTexture(0, tex_column_indices, g.d_column_indices, sizeof(VertexId) * g.m), 
        "CsrGraph: bindTexture(d_column_indices) failed", __FILE__, __LINE__)) exit(1);        

    if (util::handleError(hipBindTexture(0, tex_row_offsets, g.d_row_offsets, sizeof(SizeT) * (g.n + 1)), 
        "CsrGraph: bindTexture(d_row_offsets) failed", __FILE__, __LINE__)) exit(1);
        
    printf("Done texture memory binding.\n");


    while (worksetSize > 0) {

        // kick off timer first
        util::GpuTimer gpu_timer;
        gpu_timer.start();

        lastWorksetSize = worksetSize;

        // spawn minimal(but enough) software blocks to cover the workset
        blockNum = (worksetSize * mapping_factor % block_size == 0 ? 
            worksetSize * mapping_factor / block_size :
            worksetSize * mapping_factor / block_size + 1);
        
        // safe belt: grid width has a limit of 65535
        if (blockNum > 65535) blockNum = 65535;


        if (warp_mapped) {
            if (unordered) {
                BFSKernel_queue_group_map<VertexId, SizeT, Value, false><<<blockNum, block_size>>>(
                    g.d_row_offsets,
                    g.d_column_indices,
                    workset[selector].d_elems,
                    workset[selector].d_sizep,
                    workset[selector ^ 1].d_elems,
                    workset[selector ^ 1].d_sizep,
                    levels.d_elems,
                    curLevel,     
                    visited.d_elems,
                    group_size,
                    group_per_block);

            } else {
                // unorderd
                BFSKernel_queue_group_map<VertexId, SizeT, Value, true><<<blockNum, block_size>>>(
                    g.d_row_offsets,
                    g.d_column_indices,
                    workset[selector].d_elems,
                    workset[selector].d_sizep,
                    workset[selector ^ 1].d_elems,
                    workset[selector ^ 1].d_sizep,
                    levels.d_elems,
                    curLevel,     
                    visited.d_elems,
                    group_size,
                    group_per_block);
            }

        } else { // thread map

            if (unordered) {
                BFSKernel_queue_thread_map<VertexId, SizeT, Value, false><<<blockNum, block_size>>>(
                    g.d_row_offsets,                                        
                    g.d_column_indices,
                    workset[selector].d_elems,
                    workset[selector].d_sizep,
                    workset[selector ^ 1].d_elems,
                    workset[selector ^ 1].d_sizep,
                    levels.d_elems,
                    curLevel,     
                    visited.d_elems);

            } else {
                // unordered
                BFSKernel_queue_thread_map<VertexId, SizeT, Value, true><<<blockNum, block_size>>>(
                    g.d_row_offsets,                                        
                    g.d_column_indices,
                    workset[selector].d_elems,
                    workset[selector].d_sizep,
                    workset[selector ^ 1].d_elems,
                    workset[selector ^ 1].d_sizep,
                    levels.d_elems,
                    curLevel,     
                    visited.d_elems);
            }
        }


        if (util::handleError(hipDeviceSynchronize(), "BFSKernel failed ", __FILE__, __LINE__)) break;
        worksetSize = workset[selector ^ 1].size();

        gpu_timer.stop();

        if (instrument) printf("%d\t%d\t%d\t%f\n", curLevel, lastWorksetSize, blockNum, gpu_timer.elapsedMillis());        


        total_milllis += gpu_timer.elapsedMillis();
        accumulatedBlocks += blockNum;
        curLevel += 1;
        selector = selector ^ 1;

    } // endwhile


    
    printf("GPU queued bfs terminates\n");  
    float billion_edges_per_second = (float)g.m / total_milllis / 1000000.0;
    printf("Time(s):\t%f\nSpeed(BE/s):\t%f\n", total_milllis / 1000.0, billion_edges_per_second);
    printf("Accumulated Blocks: \t%d\n", accumulatedBlocks);

    levels.print_log();

    levels.del();
    visited.del();
    workset[0].del();
    workset[1].del();
    
}


} // BFS

} // Morgen